﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <stdlib.h>
#include <stdint.h>
#include <iostream>
#include <chrono>

#include "Worker.cuh"

#include "lib/Int.h"
#include "lib/Math.cuh"
#include "lib/util.h"
#include "lib/SECP256k1.h"

#include "bloom_filter.hpp"

Secp256K1* secp;

using namespace std;

bool readArgs(int argc, char** argv);
void prepareAlphabet();
bool checkDevice();
void processCandidate(Int& toTest);
bool read_file(const std::string& file_name);
void incrementBase58(int count, int* key);
void printSpeed(double speed);
void saveStatus();
hipError_t processCuda();

int DEVICE_NR = 0;
unsigned int BLOCK_THREADS = 0;
unsigned int BLOCK_NUMBER = 0;
unsigned int THREAD_STEPS = 5;
const int ALPHABET_LEN = 57;
char ALPHABET[ALPHABET_LEN];

string fileResult = "result.txt";
string fileStatus = "fileStatus.txt";
string fileInput = "";
int fileStatusInterval = 60;

const int KEY_LENGTH = 21;
int KEY_START_IX[KEY_LENGTH];
string KEY_START = "";
uint64_t outputSize;

bloom_parameters parameters;
bloom_filter filter;


int main(int argc, char** argv)
{
    prepareAlphabet();
    parameters.false_positive_probability = 0.000000001;
    parameters.projected_element_count = 5000;
    parameters.compute_optimal_parameters();
    filter = bloom_filter(parameters);

    readArgs(argc, argv);
    if (!checkDevice()) {
        return -1;
    }

    secp = new Secp256K1();
    secp->Init();

    auto time = std::chrono::system_clock::now();
    std::time_t s_time = std::chrono::system_clock::to_time_t(time);
    std::cout << "Work started at " << std::ctime(&s_time);

    hipError_t cudaStatus = processCuda();

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Device reset failed!");
        return 1;
    }
    
    return 0;
}

hipError_t processCuda() {
    hipError_t cudaStatus;
    int buffStart[KEY_LENGTH];
    int *dev_buffStart = new int[KEY_LENGTH];

    int COLLECTOR_SIZE = BLOCK_NUMBER;
    unsigned int* buffAlphabet = new unsigned int[ALPHABET_LEN];
    for (int i = 0; i < ALPHABET_LEN; i++) {
        buffAlphabet[i] = ALPHABET[i];
    }
    cudaStatus = loadAlphabet(buffAlphabet);

    unsigned int* buffDeviceResult = new unsigned int[outputSize * 8];
    unsigned int* dev_buffDeviceResult = new unsigned int[outputSize * 8];
    for (int i = 0; i < outputSize * 8; i++) {
        buffDeviceResult[i] = 0;
    }
    cudaStatus = hipMalloc((void**)&dev_buffDeviceResult, outputSize * 8 * sizeof(unsigned int));
    cudaStatus = hipMemcpyAsync(dev_buffDeviceResult, buffDeviceResult, outputSize * 8 * sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int* buffResult = new unsigned int[COLLECTOR_SIZE * 8];
    unsigned int* dev_buffResult = new unsigned int[COLLECTOR_SIZE * 8];
    cudaStatus = hipMalloc((void**)&dev_buffResult, COLLECTOR_SIZE * 8 * sizeof(unsigned int));
    cudaStatus = hipMemcpy(dev_buffResult, buffResult, COLLECTOR_SIZE * 8 * sizeof(unsigned int), hipMemcpyHostToDevice);

    bool* buffCollectorWork = new bool[1];
    buffCollectorWork[0] = false;
    bool* dev_buffCollectorWork = new bool[1];
    cudaStatus = hipMalloc((void**)&dev_buffCollectorWork, 1 * sizeof(bool));
    cudaStatus = hipMemcpy(dev_buffCollectorWork, buffCollectorWork, 1 * sizeof(bool), hipMemcpyHostToDevice);

    cudaStatus = hipMalloc((void**)&dev_buffStart, KEY_LENGTH * sizeof(int));

    uint64_t counter = 0;
    std::chrono::steady_clock::time_point beginCountHashrate = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point beginCountStatus = std::chrono::steady_clock::now();

    while (true) {
        for (int i = 0; i < KEY_LENGTH; i++) {
            buffStart[i] = KEY_START_IX[i];
        }
        cudaStatus = hipMemcpy(dev_buffStart, buffStart, KEY_LENGTH * sizeof(int), hipMemcpyHostToDevice);
        kernelMinikeys <<<BLOCK_NUMBER, BLOCK_THREADS >>> (dev_buffDeviceResult, dev_buffCollectorWork, dev_buffStart, THREAD_STEPS);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy(buffCollectorWork, dev_buffCollectorWork, 1 * sizeof(bool), hipMemcpyDeviceToHost);
        bool anyResult = buffCollectorWork[0];
        buffCollectorWork[0] = false;
        cudaStatus = hipMemcpyAsync(dev_buffCollectorWork, buffCollectorWork, 1 * sizeof(bool), hipMemcpyHostToDevice);
        while (anyResult) {
            resultCollector << <BLOCK_NUMBER, 1 >> > (dev_buffDeviceResult, dev_buffResult, THREAD_STEPS * BLOCK_THREADS);
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
                goto Error;
            }
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
                goto Error;
            }            
            cudaStatus = hipMemcpy(buffResult, dev_buffResult, COLLECTOR_SIZE * 8 * sizeof(unsigned int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }
            anyResult = false;
            for (int resultIx = 0; resultIx < COLLECTOR_SIZE * 8;) {
                if (buffResult[resultIx] != 0x0 || buffResult[resultIx + 1] != 0x0 || buffResult[resultIx + 2] != 0x0 || buffResult[resultIx + 3] != 0x0 ||
                    buffResult[resultIx + 4] != 0x0 || buffResult[resultIx + 5] != 0x0 || buffResult[resultIx + 6] != 0x0 || buffResult[resultIx + 7] != 0x0) {
                    Int toTest = new Int();
                    for (int b = 0; b < 8; b++) {
                        toTest.bits[7-b] = buffResult[resultIx + b];
                    }
                    toTest.bits[8] = 0x0;
                    toTest.bits[9] = 0x0;
                    processCandidate(toTest);
                    anyResult = true;
                }
                resultIx += 8;
            }
        }
        incrementBase58(outputSize, KEY_START_IX);
        counter += outputSize;
        int64_t tHash = std::chrono::duration_cast<std::chrono::seconds>(std::chrono::steady_clock::now() - beginCountHashrate).count();
        int64_t tStatus = std::chrono::duration_cast<std::chrono::seconds>(std::chrono::steady_clock::now() - beginCountStatus).count();
        if (tHash > 5) {
            double speed = (double)((double)counter / tHash) / 1000000.0;
            printSpeed(speed);
            counter = 0;
            beginCountHashrate = std::chrono::steady_clock::now();
        }
        if (tStatus > fileStatusInterval) {
            saveStatus();
            beginCountStatus = std::chrono::steady_clock::now();
        }
    }
Error:
    hipFree(dev_buffResult);
    hipFree(dev_buffDeviceResult);
    hipFree(dev_buffStart);
    hipFree(dev_buffCollectorWork);
    return cudaStatus;
}

void processCandidate(Int& toTest) {
    FILE* keys;
    char rmdhash[21], address[50];
    Point publickey = secp->ComputePublicKey(&toTest);
    secp->GetHash160(P2PKH, false, publickey, (unsigned char*)rmdhash);
    addressToBase58(rmdhash, address);
    string a = address;
    if (filter.contains(a)) {
        printf("found: %s - %s\n", address, toTest.GetBase16().c_str());
        keys = fopen(fileResult.c_str(), "a+");
        fprintf(keys, "%s\n", address);
        fprintf(keys, "%s\n\n", toTest.GetBase16().c_str());
        fclose(keys);
    }
}

void prepareAlphabet() {
    string a = "23456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";
    for (int i = 0; i < ALPHABET_LEN; i++) {
        ALPHABET[i] = a.at(i);
    }
}

bool readArgs(int argc, char** argv) {
    int a = 1;
    while (a < argc) {
        if (strcmp(argv[a], "-d") == 0) {
            a++;
            DEVICE_NR = strtol(argv[a], NULL, 10);
        }
        else if (strcmp(argv[a], "-t") == 0) {
            a++;
            BLOCK_THREADS = strtol(argv[a], NULL, 10);
        }
        else if (strcmp(argv[a], "-b") == 0) {
            a++;
            BLOCK_NUMBER = strtol(argv[a], NULL, 10);
        }
        else if (strcmp(argv[a], "-input") == 0) {
            a++;
            fileInput = string(argv[a]);
            read_file(fileInput);
        }
        else if (strcmp(argv[a], "-rangeStart") == 0) {
            a++;
            KEY_START= string(argv[a]);
            while (KEY_START.length() < KEY_LENGTH+1) {
                KEY_START.append("1");
            }
            for (int i = 1, ix=0; i < KEY_LENGTH+1; i++, ix++) {
                for (int c = 0; c < 58; c++) {
                    if (ALPHABET[c] == KEY_START.at(i)) {
                        KEY_START_IX[ix] = c;
                        break;
                    }
                }
            }
        }
        a++;
    }
    return true;
}

bool checkDevice() {
    hipError_t cudaStatus = hipSetDevice(DEVICE_NR);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "device %d failed!", DEVICE_NR);
        return false;
    }
    else {
        hipDeviceProp_t props;
        cudaStatus = hipGetDeviceProperties(&props, DEVICE_NR);
        printf("Using:\n");
        printf("%s (%2d procs)\n", props.name, props.multiProcessorCount);
        printf("maxThreadsPerBlock: %2d\n\n", props.maxThreadsPerBlock);
        if (BLOCK_NUMBER == 0) {
            BLOCK_NUMBER = props.multiProcessorCount * 8;
        }
        if (BLOCK_THREADS == 0) {
            BLOCK_THREADS = (props.maxThreadsPerBlock / 8) * 5;
        }
        outputSize = (uint64_t) BLOCK_NUMBER * BLOCK_THREADS * THREAD_STEPS;

    }
    return true;
}

bool read_file(const std::string& file_name) {
    std::ifstream stream(file_name.c_str());

    if (!stream)
    {
        std::cout << "Error: Failed to open file '" << file_name << "'" << std::endl;
        return false;
    }
    std::string buffer;

    while (std::getline(stream, buffer))
    {
        filter.insert(buffer);
    }
    printf("loaded: %d\n", filter.element_count());
}

void incrementBase58(int count, int* key) {    
    for (int c = 0; c < count; c++) {
        int i = 20;
        do {
            key[i] = (key[i] + 1) % 57;
        } while (key[i--] == 0 && i >= 0);
    }
}

void saveStatus() {
    FILE* stat = fopen(fileStatus.c_str(), "w");
    auto time = std::chrono::system_clock::now();
    std::time_t s_time = std::chrono::system_clock::to_time_t(time);
    fprintf(stat, "%s\n", std::ctime(&s_time));
    string key="";
    for (int i = 0; i < KEY_LENGTH; i++) {
        key+=ALPHABET[KEY_START_IX[i]];        
    }
    fprintf(stat, "-rangeStart=S%s\n", key.c_str());
    fclose(stat);
}

void printSpeed(double speed) {
    std::string speedStr;
    if (speed < 0.01) {
        speedStr = "< 0.01 MKey/s";
    }
    else {
        if (speed < 1000) {
            speedStr = formatDouble("%.3f", speed) + " MKey/s";
        }
        else {
            speed /= 1000;
            if (speed < 1000) {
                speedStr = formatDouble("%.3f", speed) + " GKey/s";
            }
            else {
                speed /= 1000;
                speedStr = formatDouble("%.3f", speed) + " TKey/s";
            }
        }
    }
    printf("\r %s   ", speedStr.c_str()); 
    fflush(stdout);
}

/*
* test : SkK5VPtmTm3mQKYaJQFRZP
* 1PzEGi7a6UEGCAXtGjZj8kBX2VEHcLMrqd - F30C1DDD12EA91BD35D5D1B83EAC611717D99DA826F207C3C3D4839E271648CB
*/